#include <hip/hip_runtime.h>
#include <stdio.h>

namespace jrb::infrastructure::cuda {

__global__ void helloFromGPU() {
  printf("Hello World from GPU! Thread ID: %d, Block ID: %d\n", threadIdx.x, blockIdx.x);
}

// Pure CUDA function without spdlog dependencies
extern "C" hipError_t cuda_launch_hello_kernel() {
  printf("Hello World from CPU!\n");

  helloFromGPU<<<64, 256>>>();
  hipError_t error = hipDeviceSynchronize();

  if (error != hipSuccess) {
    return error;
  }

  printf("Program completed successfully!\n");
  return hipSuccess;
}

}  // namespace jrb::infrastructure::cuda
