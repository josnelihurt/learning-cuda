#include "hip/hip_runtime.h"
#include <stdio.h>
#include "infrastructure/cuda/simple_kernel.h"

namespace jrb::infrastructure::cuda {

__global__ void helloFromGPU() {
    printf("Hello World from GPU! Thread ID: %d, Block ID: %d\n", 
           threadIdx.x, blockIdx.x);
}

void launch_hello_kernel() {
    printf("Hello World from CPU!\n");
    
    helloFromGPU<<<64, 256>>>();
    hipDeviceSynchronize();
    
    printf("Program completed successfully!\n");
}

}  // namespace jrb::infrastructure::cuda
